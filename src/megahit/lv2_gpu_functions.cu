#include "hip/hip_runtime.h"
/*
 *  MEGAHIT
 *  Copyright (C) 2014 - 2015 The University of Hong Kong & L3 Bioinformatics Limited
 *
 *  This program is free software: you can redistribute it and/or modify
 *  it under the terms of the GNU General Public License as published by
 *  the Free Software Foundation, either version 3 of the License, or
 *  (at your option) any later version.
 *
 *  This program is distributed in the hope that it will be useful,
 *  but WITHOUT ANY WARRANTY; without even the implied warranty of
 *  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 *  GNU General Public License for more details.
 *
 *  You should have received a copy of the GNU General Public License
 *  along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

/* contact: Dinghua Li <dhli@cs.hku.hk> */

#include "lv2_gpu_functions.h"
#include <stdio.h>
#include <assert.h>
#include "cub/util_allocator.cuh"
#include "cub/device/device_radix_sort.cuh"
#include "utils.h"

using namespace hipcub;

static CachingDeviceAllocator g_allocator(true);
const int kGPUThreadPerBlock = 256;

void cuda_init() {
    hipFree(0);
}

void get_cuda_memory(size_t &free_mem, size_t &total_mem) {
    assert(hipMemGetInfo(&free_mem, &total_mem) == hipSuccess);
}

void alloc_gpu_buffers(void* &gpu_key_buffer1,
                       void* &gpu_key_buffer2,
                       void* &gpu_value_buffer1,
                       void* &gpu_value_buffer2,
                       size_t max_num_items) {
    CubDebugExit(g_allocator.DeviceAllocate((void**)&gpu_key_buffer1, sizeof(uint32_t) * max_num_items));
    CubDebugExit(g_allocator.DeviceAllocate((void**)&gpu_key_buffer2, sizeof(uint32_t) * max_num_items));
    CubDebugExit(g_allocator.DeviceAllocate((void**)&gpu_value_buffer1, sizeof(uint32_t) * max_num_items));
    CubDebugExit(g_allocator.DeviceAllocate((void**)&gpu_value_buffer2, sizeof(uint32_t) * max_num_items));
}

void free_gpu_buffers(void* gpu_key_buffer1,
                      void* gpu_key_buffer2,
                      void* gpu_value_buffer1,
                      void* gpu_value_buffer2) {
    // free device memory
    CubDebugExit(g_allocator.DeviceFree(gpu_key_buffer1));
    CubDebugExit(g_allocator.DeviceFree(gpu_key_buffer2));
    CubDebugExit(g_allocator.DeviceFree(gpu_value_buffer1));
    CubDebugExit(g_allocator.DeviceFree(gpu_value_buffer2));
}


// device function for permuting an array
__global__ void permutation_kernel(uint32_t *index, uint32_t *val, uint32_t *new_val, uint32_t num_items) {
    int tid = blockIdx.x * kGPUThreadPerBlock + threadIdx.x;
    if (tid < num_items)
        new_val[tid] = val[index[tid]];
}

// device function for reset permutation
__global__ void reset_permutation_kernel(uint32_t *permutation, uint32_t num_items) {
    int tid = blockIdx.x * kGPUThreadPerBlock + threadIdx.x;
    if (tid < num_items)
        permutation[tid] = tid;
}

// single thread
void lv2_gpu_sort(uint32_t *lv2_substrings,
                  uint32_t *permutation,
                  int words_per_substring,
                  int64_t lv2_num_items,
                  void* gpu_key_buffer1,
                  void* gpu_key_buffer2,
                  void* gpu_value_buffer1,
                  void* gpu_value_buffer2) {
    DoubleBuffer<uint32_t> d_keys;
    DoubleBuffer<uint32_t> d_values;
    d_keys.d_buffers[0] = static_cast<__typeof(d_keys.d_buffers[0])>(gpu_key_buffer1);
    d_keys.d_buffers[1] = static_cast<__typeof(d_keys.d_buffers[1])>(gpu_key_buffer2);
    d_values.d_buffers[0] = static_cast<__typeof(d_values.d_buffers[0])>(gpu_value_buffer1);
    d_values.d_buffers[1] = static_cast<__typeof(d_values.d_buffers[1])>(gpu_value_buffer2);

    // Initialize permutation array
    int num_gpu_blocks = DivCeiling(lv2_num_items, kGPUThreadPerBlock);
    reset_permutation_kernel<<<num_gpu_blocks, kGPUThreadPerBlock>>>(d_values.d_buffers[d_values.selector], lv2_num_items);

    // Allocate temporary storage
    size_t  temp_storage_bytes  = 0;
    void *gpu_temp_storage     = NULL;
    CubDebugExit(DeviceRadixSort::SortPairs(gpu_temp_storage, temp_storage_bytes, d_keys, d_values, lv2_num_items));
    CubDebugExit(g_allocator.DeviceAllocate(&gpu_temp_storage, temp_storage_bytes));

    for (int64_t iteration = words_per_substring - 1; iteration >= 0; --iteration) {
        if (iteration == words_per_substring - 1) { // first iteration
            CubDebugExit(hipMemcpy(d_keys.d_buffers[d_keys.selector], lv2_substrings + (iteration * lv2_num_items),
                                    sizeof(uint32_t) * lv2_num_items, hipMemcpyHostToDevice));
        } else {
            CubDebugExit(hipMemcpy(d_keys.d_buffers[1 - d_keys.selector], lv2_substrings + (iteration * lv2_num_items),
                                    sizeof(uint32_t) * lv2_num_items, hipMemcpyHostToDevice));

            permutation_kernel<<<num_gpu_blocks, kGPUThreadPerBlock>>>(d_values.d_buffers[d_values.selector],
                    d_keys.d_buffers[1 - d_keys.selector], d_keys.d_buffers[d_keys.selector],
                    lv2_num_items);
        }

        // Run
        CubDebugExit(DeviceRadixSort::SortPairs(gpu_temp_storage, temp_storage_bytes, d_keys, d_values, lv2_num_items));
    }

    // copy answer back to host
    CubDebugExit(hipMemcpy(permutation, d_values.d_buffers[d_values.selector], sizeof(uint32_t) * lv2_num_items, hipMemcpyDeviceToHost));

    CubDebugExit(g_allocator.DeviceFree(gpu_temp_storage));
}